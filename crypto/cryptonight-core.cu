#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <string.h>
#include <sys/time.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#if CUDA_VERSION >= 9000 && __CUDA_ARCH__ >= 300
#undef __shfl
#define __shfl(var, srcLane, width) __shfl_sync(0xFFFFFFFFu, var, srcLane, width)
#endif

#include "cryptonight.h"

#define LONG_SHL32 19 // 1<<19 (uint32_t* index)
#define LONG_SHL64 18 // 1<<18 (uint64_t* index)
#define LONG_LOOPS32 0x80000U

#ifndef _WIN32
#include <unistd.h>
#endif

#include "cn_aes.cuh"

__global__
void cryptonight_gpu_phase1(const uint32_t threads, uint32_t * __restrict__ d_long_state,
	uint32_t * __restrict__ ctx_state, uint32_t * __restrict__ ctx_key1)
{
	__shared__ uint32_t sharedMemory[1024];

	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x) >> 3;
	if(thread < threads)
	{
		cn_aes_gpu_init(sharedMemory);
		__syncthreads();

		const int sub = (threadIdx.x & 7) << 2;
		uint32_t *longstate = &d_long_state[(thread << LONG_SHL32) + sub];

		uint32_t key[40], text[4];

		MEMCPY8(key, ctx_key1 + thread * 40, 20);
		MEMCPY8(text, ctx_state + thread * 50 + sub + 16, 2);

		for(int i = 0; i < LONG_LOOPS32; i += 32)
		{
			cn_aes_pseudo_round_mut(sharedMemory, text, key);
			MEMCPY8(&longstate[i], text, 2);
		}
	}
}

// --------------------------------------------------------------------------------------------------------------

__device__ __forceinline__ ulonglong2 cuda_mul128(const uint64_t multiplier, const uint64_t multiplicand)
{
	ulonglong2 product;
	product.x = __umul64hi(multiplier, multiplicand);
	product.y = multiplier * multiplicand;
	return product;
}

static __forceinline__ __device__ void operator += (ulonglong2 &a, const ulonglong2 b) {
	a.x += b.x; a.y += b.y;
}

static __forceinline__ __device__ ulonglong2 operator ^ (const ulonglong2 &a, const ulonglong2 &b) {
	return make_ulonglong2(a.x ^ b.x, a.y ^ b.y);
}

__device__ __forceinline__ void MUL_SUM_XOR_DST_0(const uint64_t m, uint4 &a, void* far_dst)
{
	ulonglong2 d = AS_UL2(far_dst);
	ulonglong2 p = cuda_mul128(m, d.x);
	p += AS_UL2(&a);
	AS_UL2(&a) = p ^ d;
	AS_UL2(far_dst) = p;
}

__global__
#if __CUDA_ARCH__ >= 500
//__launch_bounds__(128,12) /* force 40 regs to allow -l ...x32 */
#endif
void cryptonight_gpu_phase2(const uint32_t threads, const uint16_t bfactor, const uint32_t partidx,
	uint64_t * __restrict__ d_long_state, uint32_t * __restrict__ d_ctx_a, uint32_t * __restrict__ d_ctx_b)
{
	__shared__ __align__(16) uint32_t sharedMemory[1024];
	cn_aes_gpu_init(sharedMemory);
	__syncthreads();

	const uint32_t thread = blockDim.x * blockIdx.x + threadIdx.x;

	if (thread < threads)
	{
		const uint32_t batchsize = ITER >> (2 + bfactor);
		const uint32_t start = partidx * batchsize;
		const uint32_t end = start + batchsize;

		void * ctx_a = (void*)(&d_ctx_a[thread << 2U]);
		void * ctx_b = (void*)(&d_ctx_b[thread << 2U]);
		uint4 A = AS_UINT4(ctx_a); // ld.global.u32.v4
		uint4 B = AS_UINT4(ctx_b);

		uint64_t * long_state = &d_long_state[thread << LONG_SHL64];
		for (int i = start; i < end; i++) // end = 262144
		{
			uint4 C;

			uint32_t j = (A.x & E2I_MASK) >> 3;
			cn_aes_single_round_b((uint8_t*)sharedMemory, &long_state[j], A, &C);
			AS_UINT4(&long_state[j]) = C ^ B; // st.global.u32.v4
			MUL_SUM_XOR_DST_0((AS_UL2(&C)).x, A, &long_state[(C.x & E2I_MASK) >> 3]);

			j = (A.x & E2I_MASK) >> 3;
			cn_aes_single_round_b((uint8_t*)sharedMemory, &long_state[j], A, &B);
			AS_UINT4(&long_state[j]) = C ^ B;
			MUL_SUM_XOR_DST_0((AS_UL2(&B)).x, A, &long_state[(B.x & E2I_MASK) >> 3]);
		}

		if (bfactor) {
			AS_UINT4(ctx_a) = A;
			AS_UINT4(ctx_b) = B;
		}
	}
}

// --------------------------------------------------------------------------------------------------------------

#if 0
#if UINTPTR_MAX == UINT64_MAX
#define LPTR "l"
#else
#define LPTR "r"
#endif

__device__ __forceinline__ uint64_t loadGlobal64(uint64_t * const addr) {
	uint64_t x;
	asm volatile("ld.global.cg.u64 %0, [%1];" : "=l"(x) : LPTR (addr));
	return x;
}

__device__ __forceinline__ uint32_t loadGlobal32(uint32_t * const addr) {
	uint32_t x;
	asm volatile("ld.global.cg.u32 %0, [%1];" : "=r"(x) : LPTR (addr));
	return x;
}

__device__ __forceinline__ void storeGlobal32(uint32_t* addr, uint32_t const & val) {
	asm volatile("st.global.cg.u32 [%0], %1;" : : LPTR (addr), "r"(val));
}

__device__ __forceinline__ uint32_t variant1_1(const uint32_t src)
{
	const uint32_t tmp = (src >> 24); // __byte_perm(src, 0, 0x7773);
	const uint32_t index = (((tmp >> 3) & 6u) | (tmp & 1u)) << 1;
	return (src & 0x00ffffffu) | ((tmp ^ ((0x75310u >> index) & 0x30u)) << 24);
}

__global__
void monero_phase2_messed(const uint32_t threads, const uint32_t bfactor, const uint32_t partidx,
	uint32_t * __restrict__ d_long_state, uint32_t * __restrict__ d_ctx_a, uint32_t * __restrict__ d_ctx_b,
	uint64_t * __restrict__ d_tweak)
{
	__shared__ uint32_t sharedMemory[1024];
	cn_aes_gpu_init(sharedMemory);
	__syncthreads();
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x) >> 2;
	if (thread >= threads)
		return;

	const uint32_t batchsize = ITER >> (2U + bfactor);
	const uint32_t start = partidx * batchsize;
	const uint32_t end = start + batchsize;

	const uint32_t subthr = threadIdx.x & 3;
	const uint32_t thrctx = thread << 2;
	uint32_t * ctx_a = &d_ctx_a[thrctx];
	uint32_t * ctx_b = &d_ctx_b[thrctx];
	uint32_t * long_state = &d_long_state[thread << LONG_SHL32];

	uint32_t d[2], t1[2], t2[2];
	uint32_t a = ctx_a[subthr];
	d[1] = ctx_b[subthr];

	uint32_t tweak[2];
	AS_UINT2(&tweak) = AS_UINT2(&d_tweak[thread]);

	for (uint32_t i = start; i < end; i++)
	{
		#pragma unroll 2
		for (int x = 0; x < 2; x++)
		{
			uint32_t j = ((__shfl(a, 0, 4) & 0x1FFFF0) >> 2) + subthr;

			const uint32_t x_0 = loadGlobal32<uint32_t>(long_state + j);
			const uint32_t x_1 = __shfl(x_0, subthr + 1, 4);
			const uint32_t x_2 = __shfl(x_0, subthr + 2, 4);
			const uint32_t x_3 = __shfl(x_0, subthr + 3, 4);

			// t_fn = aes shared mem read
			d[x] = a ^ 	t_fn0(x_0 & 0xff) ^
				t_fn1((x_1 >>  8) & 0xff) ^
				t_fn2((x_2 >> 16) & 0xff) ^
				t_fn3((x_3 >> 24));
			t1[0] = __shfl(d[x], 0, 4);

			uint32_t z = d[0] ^ d[1];
			if (subthr == 2) z = variant1_1(z);
			storeGlobal32(long_state + j, z);

			// -----------------------------------------------------------
			j = ((*t1 & 0x1FFFF0) >> 2) + subthr;

			uint32_t yy[2];
			AS_U64(yy) = loadGlobal64<uint64_t>(((uint64_t *)long_state) + (j >> 1));

			t1[1] = __shfl(d[x], 1, 4);

			uint32_t sub2 = (threadIdx.x & 2);
			t2[0] = __shfl(a, sub2, 4);
			t2[1] = __shfl(a, sub2 + 1U, 4);

			uint32_t zz[2];
			zz[0] = __shfl(yy[0], 0, 4);
			zz[1] = __shfl(yy[1], 0, 4);

			AS_U64(t2) += sub2 ? (AS_U64(t1) * AS_U64(zz)) : __umul64hi(AS_U64(t1), AS_U64(zz));

			uint32_t s1 = subthr & 1U;
			z = AS_U64(t2) >> (s1 * 32U); // hi or lo dword
			//z = __byte_perm(t2[0], t2[1], s1 ? 0x7654 : 0x3210);
			storeGlobal32(long_state + j, sub2 ? tweak[s1] ^ z : z);
			a = z ^ yy[s1];
		}
	}

	if (bfactor) {
		ctx_a[subthr] = a;
		ctx_b[subthr] = d[1];
	}
}
#endif

__device__ __forceinline__ void store_variant1(uint64_t* long_state, uint4 Z)
{
	const uint32_t tmp = (Z.z >> 24); // __byte_perm(src, 0, 0x7773);
	const uint32_t index = (((tmp >> 3) & 6u) | (tmp & 1u)) << 1;
	Z.z = (Z.z & 0x00ffffffu) | ((tmp ^ ((0x75310u >> index) & 0x30u)) << 24);
	AS_UINT4(long_state) = Z;
}

__device__ __forceinline__ void MUL_SUM_XOR_DST_1(const uint64_t m, uint4 &a, void* far_dst, uint64_t tweak)
{
	ulonglong2 d = AS_UL2(far_dst);
	ulonglong2 p = cuda_mul128(m, d.x);
	p += AS_UL2(&a);
	AS_UL2(&a) = p ^ d;
	p.y = p.y ^ tweak;
	AS_UL2(far_dst) = p;
}

__global__
void monero_gpu_phase2(const uint32_t threads, const uint16_t bfactor, const uint32_t partidx,
	uint64_t * __restrict__ d_long_state, uint32_t * __restrict__ d_ctx_a, uint32_t * __restrict__ d_ctx_b,
	uint64_t * __restrict__ d_tweak)
{
	__shared__ __align__(16) uint32_t sharedMemory[1024];
	cn_aes_gpu_init(sharedMemory);
	__syncthreads();

	const uint32_t thread = blockDim.x * blockIdx.x + threadIdx.x;

	if (thread < threads)
	{
		const uint32_t batchsize = ITER >> (2 + bfactor);
		const uint32_t start = partidx * batchsize;
		const uint32_t end = start + batchsize;

		uint64_t tweak = d_tweak[thread];

		void * ctx_a = (void*)(&d_ctx_a[thread << 2]);
		void * ctx_b = (void*)(&d_ctx_b[thread << 2]);
		uint4 A = AS_UINT4(ctx_a); // ld.global.u32.v4
		uint4 B = AS_UINT4(ctx_b);

		uint64_t * long_state = &d_long_state[thread << LONG_SHL64];
		for (int i = start; i < end; i++) // end = 262144
		{
			uint4 C;

			uint32_t j = (A.x & E2I_MASK) >> 3;
			cn_aes_single_round_b((uint8_t*)sharedMemory, &long_state[j], A, &C);
			store_variant1(&long_state[j], C ^ B); // st.global.u32.v4
			MUL_SUM_XOR_DST_1((AS_UL2(&C)).x, A, &long_state[(C.x & E2I_MASK) >> 3], tweak);

			j = (A.x & E2I_MASK) >> 3;
			cn_aes_single_round_b((uint8_t*)sharedMemory, &long_state[j], A, &B);
			store_variant1(&long_state[j], C ^ B); // st.global.u32.v4
			MUL_SUM_XOR_DST_1((AS_UL2(&B)).x, A, &long_state[(B.x & E2I_MASK) >> 3], tweak);
		}

		if (bfactor) {
			AS_UINT4(ctx_a) = A;
			AS_UINT4(ctx_b) = B;
		}
	}
}

// --------------------------------------------------------------------------------------------------------------

__global__
void cryptonight_gpu_phase3(const uint32_t threads, const uint32_t * __restrict__ d_long_state,
	uint32_t * __restrict__ d_ctx_state, const uint32_t * __restrict__ d_ctx_key2)
{
	__shared__ uint32_t sharedMemory[1024];

	cn_aes_gpu_init(sharedMemory);
	__syncthreads();

	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x) >> 3;

	if(thread < threads)
	{
		const int sub = (threadIdx.x & 7) << 2;
		const uint32_t *longstate = &d_long_state[(thread << LONG_SHL32) + sub];
		uint32_t key[40], text[4];
		MEMCPY8(key, d_ctx_key2 + thread * 40, 20);
		MEMCPY8(text, d_ctx_state + thread * 50 + sub + 16, 2);

		for(int i = 0; i < LONG_LOOPS32; i += 32)
		{
			#pragma unroll
			for(int j = 0; j < 4; ++j)
				text[j] ^= longstate[i + j];

			cn_aes_pseudo_round_mut(sharedMemory, text, key);
		}

		MEMCPY8(d_ctx_state + thread * 50 + sub + 16, text, 2);
	}
}

// --------------------------------------------------------------------------------------------------------------

extern int device_bfactor[MAX_GPUS];

__host__
void cryptonight_core_cuda(int thr_id, int blocks, int threads, uint64_t *d_long_state, uint32_t *d_ctx_state,
	uint32_t *d_ctx_a, uint32_t *d_ctx_b, uint32_t *d_ctx_key1, uint32_t *d_ctx_key2, int variant, uint64_t *d_ctx_tweak)
{
	dim3 grid(blocks);
	dim3 block(threads);
	dim3 block4(threads << 2);
	dim3 block8(threads << 3);

	const uint16_t bfactor = (uint16_t) device_bfactor[thr_id];
	const uint32_t partcount = 1U << bfactor;
	const uint32_t throughput = (uint32_t) (blocks*threads);

	const int bsleep = bfactor ? 100 : 0;
	const int dev_id = device_map[thr_id];

	cryptonight_gpu_phase1 <<<grid, block8>>> (throughput, (uint32_t*) d_long_state, d_ctx_state, d_ctx_key1);
	exit_if_cudaerror(thr_id, __FUNCTION__, __LINE__);
	if(partcount > 1) usleep(bsleep);

	for (uint32_t i = 0; i < partcount; i++)
	{
		dim3 b = device_sm[dev_id] >= 300 ? block4 : block;
		if (variant == 0)
			cryptonight_gpu_phase2 <<<grid, b>>> (throughput, bfactor, i, d_long_state, d_ctx_a, d_ctx_b);
		else
			monero_gpu_phase2 <<<grid, b>>> (throughput, bfactor, i, d_long_state, d_ctx_a, d_ctx_b, d_ctx_tweak);
			//monero_phase2_messed <<<grid, b>>> (throughput, bfactor, i, (uint32_t*) d_long_state, d_ctx_a, d_ctx_b, d_ctx_tweak);
		exit_if_cudaerror(thr_id, __FUNCTION__, __LINE__);
		if(partcount > 1) usleep(bsleep);
	}
	//hipDeviceSynchronize();
	//exit_if_cudaerror(thr_id, __FUNCTION__, __LINE__);
	cryptonight_gpu_phase3 <<<grid, block8>>> (throughput, (uint32_t*) d_long_state, d_ctx_state, d_ctx_key2);
	exit_if_cudaerror(thr_id, __FUNCTION__, __LINE__);
}
